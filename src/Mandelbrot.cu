#include "hip/hip_runtime.h"
// OpenGL Graphics includes
#include <GL/glew.h>
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#include <GL/wglew.h>
#endif
#if defined(__APPLE__) || defined(__MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// CUDA runtime
// CUDA utilities and system includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <rendercheck_gl.h>

// Includes
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cstdio>

FILE *stream ;
char g_ExecPath[300] ;

//OpenGL PBO and texture "names"
GLuint gl_PBO, gl_Tex, gl_Shader;
struct hipGraphicsResource *cuda_pbo_resource; // handles OpenGL-CUDA exchange

//Source image on the host side
uchar4 *h_Src = 0;

// Destination image on the GPU side
uchar4 *d_dst = NULL;

int imageW = 800, imageH = 600;

StopWatchInterface *hTimer = NULL;

bool haveDoubles = false;
int numSMs = 0;          // number of multiprocessors
int version = 1;             // Compute Capability

unsigned int g_TotalErrors = 0;

int *pArgc = NULL;
char **pArgv = NULL;

#define REFRESH_DELAY     10 //ms

#ifndef MAX
#define MAX(a,b) ((a > b) ? a : b)
#endif
#define BUFFER_DATA(i) ((char *)0 + i)

__global__ void ImagePass(uchar4 *dst, int imageW, int imageH)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int pixel = imageW * iy + ix;

    uchar4 color;

    color.x = 1;
    color.y = 0;
    color.z = 0;

    dst[pixel] = color;

//    dst[pixel].x = ix / imageW;
//    dst[pixel].y = iy / imageH;
//    dst[pixel].z = 1;

//    // loop until all blocks completed
//    for (unsigned int blockIndex=blockIdx.x; blockIndex < numBlocks; blockIndex += gridDim.x)
//    {
//        unsigned int blockX = blockIndex % gridWidth;
//        unsigned int blockY = blockIndex / gridWidth;
//
//        // process this block
//        const int ix = blockDim.x * blockX + threadIdx.x;
//        const int iy = blockDim.y * blockY + threadIdx.y;
//
//        if ((ix < imageW) && (iy < imageH))
//        {
//            // Calculate the location
//            const T xPos = (T)ix * scale + xOff;
//            const T yPos = (T)iy * scale + yOff;
//
//            // Calculate the Mandelbrot index for the current location
//            int m = CalcMandelbrot<T>(xPos, yPos, xJP, yJP, crunch, isJ);
//            //            int m = blockIdx.x;         // uncomment to see scheduling order
//            m = m > 0 ? crunch - m : 0;
//
//            // Convert the Mandelbrot index into a color
//            uchar4 color;
//
//            if (m)
//            {
//                m += animationFrame;
//                color.x = m * colors.x;
//                color.y = m * colors.y;
//                color.z = m * colors.z;
//            }
//            else
//            {
//                color.x = 0;
//                color.y = 0;
//                color.z = 0;
//            }
//
//            // Output the pixel
//            int pixel = imageW * iy + ix;
//
//            if (frame == 0)
//            {
//                color.w = 0;
//                dst[pixel] = color;
//            }
//            else
//            {
//                int frame1 = frame + 1;
//                int frame2 = frame1 / 2;
//                dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
//                dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
//                dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
//            }
//        }
//
//    }

} // Mandelbrot0

#define BLOCKDIM_X 16
#define BLOCKDIM_Y 16

// Increase the grid size by 1 if the image width or height does not divide evenly
// by the thread block dimensions
inline int iDivUp(int a, int b)
{
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
} // iDivUp

void renderImage()
{
	checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_dst, &num_bytes, cuda_pbo_resource));


    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

    int numWorkerBlocks = numSMs;

//    dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
//    dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

//	printf("pass\n");

	ImagePass<<<numWorkerBlocks, threads>>>(d_dst, imageW, imageH);

	hipDeviceSynchronize();

	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

//#if RUN_TIMING
//    pass = 0;
//#endif
//	float timeEstimate;
//	int startPass = pass;
//	sdkResetTimer(&hTimer);
//
//	if (bUseOpenGL)
//	{
//		// DEPRECATED: checkCudaErrors(cudaGLMapBufferObject((void**)&d_dst, gl_PBO));
//		checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
//		size_t num_bytes;
//		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_dst, &num_bytes, cuda_pbo_resource));
//	}
//
//	// Render anti-aliasing passes until we run out time (60fps approximately)
//	do
//	{
//		float xs, ys;
//
//		// Get the anti-alias sub-pixel sample location
//		GetSample(pass & 127, xs, ys);
//
//		// Get the pixel scale and offset
//		double s = scale / (float)imageW;
//		double x = (xs - (double)imageW * 0.5f) * s + xOff;
//		double y = (ys - (double)imageH * 0.5f) * s + yOff;
//
//
//		// Run the mandelbrot generator
//		if (pass && !startPass) // Use the adaptive sampling version when animating.
//			RunMandelbrot1(d_dst, imageW, imageH, crunch, x, y,
//						   xJParam, yJParam, s, colors, pass++, animationFrame, precisionMode, numSMs, g_isJuliaSet, version);
//		else
//			RunMandelbrot0(d_dst, imageW, imageH, crunch, x, y,
//						   xJParam, yJParam, s, colors, pass++, animationFrame, precisionMode, numSMs, g_isJuliaSet, version);
//
//		hipDeviceSynchronize();
//
//		// Estimate the total time of the frame if one more pass is rendered
//		timeEstimate = 0.001f * sdkGetTimerValue(&hTimer) * ((float)(pass + 1 - startPass) / (float)(pass - startPass));
//	}
//	while ((pass < 128) && (timeEstimate < 1.0f / 60.0f) && !RUN_TIMING);
//
//	if (bUseOpenGL)
//	{
//		// DEPRECATED: checkCudaErrors(cudaGLUnmapBufferObject(gl_PBO));
//		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));
//	}

}

// OpenGL display function
void displayFunc(void)
{
    sdkStartTimer(&hTimer);
//    printf("displayFunc\n");
    renderImage();

    glBindTexture(GL_TEXTURE_2D, gl_Tex);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA, GL_UNSIGNED_BYTE, BUFFER_DATA(0));

    glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, gl_Shader);
    glEnable(GL_FRAGMENT_PROGRAM_ARB);
    glDisable(GL_DEPTH_TEST);

    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f);
    glVertex2f(0.0f, 0.0f);
    glTexCoord2f(1.0f, 0.0f);
    glVertex2f(1.0f, 0.0f);
    glTexCoord2f(1.0f, 1.0f);
    glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 1.0f);
    glVertex2f(0.0f, 1.0f);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glDisable(GL_FRAGMENT_PROGRAM_ARB);

    sdkStopTimer(&hTimer);
    glutSwapBuffers();
}

void cleanup()
{
    if (h_Src)
    {
        free(h_Src);
        h_Src = 0;
    }

    sdkStopTimer(&hTimer);
    sdkDeleteTimer(&hTimer);

    //DEPRECATED: checkCudaErrors(cudaGLUnregisterBufferObject(gl_PBO));
    hipGraphicsUnregisterResource(cuda_pbo_resource);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    glDeleteBuffers(1, &gl_PBO);
    glDeleteTextures(1, &gl_Tex);
    glDeleteProgramsARB(1, &gl_Shader);
}

void initMenus() ;

void keyboardFunc(unsigned char k, int, int)
{
    switch (k)
    {
        case '\033':
        case 'q':
        case 'Q':
            printf("Shutting down...\n");

            hipDeviceReset();
            exit(EXIT_SUCCESS);
            break;

        default:
            break;
    }

}

void clickFunc(int button, int state, int x, int y)
{

}

void motionFunc(int x, int y)
{
}

void timerEvent(int value)
{
    glutPostRedisplay();
    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
}

void mainMenu(int i)
{

}

void initMenus()
{
}

// gl_Shader for displaying floating-point texture
static const char *shader_code =
    "!!ARBfp1.0\n"
    "TEX result.color, fragment.texcoord, texture[0], 2D; \n"
    "END";

GLuint compileASMShader(GLenum program_type, const char *code)
{
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB, (GLsizei) strlen(code), (GLubyte *) code);

    GLint error_pos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);

    if (error_pos != -1)
    {
        const GLubyte *error_string;
        error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Program error at position: %d\n%s\n", (int)error_pos, error_string);
        return 0;
    }

    return program_id;
}

void initOpenGLBuffers(int w, int h)
{
    // delete old buffers
    if (h_Src)
    {
        free(h_Src);
        h_Src = 0;
    }

    if (gl_Tex)
    {
        glDeleteTextures(1, &gl_Tex);
        gl_Tex = 0;
    }

    if (gl_PBO)
    {
        //DEPRECATED: checkCudaErrors(cudaGLUnregisterBufferObject(gl_PBO));
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        glDeleteBuffers(1, &gl_PBO);
        gl_PBO = 0;
    }

    // check for minimized window
    if ((w==0) && (h==0))
    {
        return;
    }

    // allocate new buffers
    h_Src = (uchar4 *)malloc(w * h * 4);

    printf("Creating GL texture...\n");
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &gl_Tex);
    glBindTexture(GL_TEXTURE_2D, gl_Tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, w, h, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_Src);
    printf("Texture created.\n");

    printf("Creating PBO...\n");
    glGenBuffers(1, &gl_PBO);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, w * h * 4, h_Src, GL_STREAM_COPY);
    //While a PBO is registered to CUDA, it can't be used
    //as the destination for OpenGL drawing calls.
    //But in our particular case OpenGL is only used
    //to display the content of the PBO, specified by CUDA kernels,
    //so we need to register/unregister it only once.

    // DEPRECATED: checkCudaErrors( cudaGLRegisterBufferObject(gl_PBO) );
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, gl_PBO,
                                                 cudaGraphicsMapFlagsWriteDiscard));
    printf("PBO created.\n");

    // load shader program
    gl_Shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);
}

void reshapeFunc(int w, int h)
{
    glViewport(0, 0, w, h);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);

    initOpenGLBuffers(w, h);
    imageW = w;
    imageH = h;
}

void initGL(int *argc, char **argv)
{
    printf("Initializing GLUT...\n");
    glutInit(argc, argv);

    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(imageW, imageH);
    glutInitWindowPosition(0, 0);
    glutCreateWindow(argv[0]);

    glutDisplayFunc(displayFunc);
    glutKeyboardFunc(keyboardFunc);
    glutMouseFunc(clickFunc);
    glutMotionFunc(motionFunc);
    glutReshapeFunc(reshapeFunc);
    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    initMenus();

    printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));

    if (!glewIsSupported("GL_VERSION_1_5 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object"))
    {
        exit(EXIT_SUCCESS);
    }

    printf("OpenGL window created.\n");
}

void initData(int argc, char **argv)
{
    // check for hardware double precision support
    int dev = 0;
    dev = findCudaDevice(argc, (const char **)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    version = deviceProp.major*10 + deviceProp.minor;

    if (version < 11)
    {
        printf("GPU compute capability is too low (1.0), program is waived\n");
        exit(EXIT_WAIVED);
    }

    haveDoubles = (version >= 13);
    numSMs = deviceProp.multiProcessorCount;

    printf("Data initialization done.\n");
}


void chooseCudaDevice(int argc, const char **argv, bool bUseOpenGL)
{
    if (bUseOpenGL)
    {
        findCudaGLDevice(argc, argv);
    }
    else
    {
        findCudaDevice(argc, argv);
    }
}

int main(int argc, char **argv)
{
    pArgc = &argc;
    pArgv = argv;

    if (checkCmdLineFlag(argc, (const char **)argv, "help"))
    {
        exit(EXIT_SUCCESS);
    }

    int mode = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "file"))
    {
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        findCudaDevice(argc, (const char **)argv); // no OpenGL usage

        // If the GPU does not meet SM1.1 capabilities, we will quit
        if (!checkCudaCapabilities(1,1))
        {
            exit(EXIT_SUCCESS);
        }

        hipDeviceReset();
        exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
    }
    else if (checkCmdLineFlag(argc, (const char **)argv, "benchmark"))
    {
        //run benchmark
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        chooseCudaDevice(argc, (const char **)argv, false); // no OpenGL usage

        // If the GPU does not meet a minimum of SM1.1 capabilities, we will quit
        if (!checkCudaCapabilities(1,1))
        {
            exit(EXIT_SUCCESS);
        }

        hipDeviceReset();
        exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
    }
    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    else if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        printf("[%s]\n", argv[0]);
        printf("   Does not explicitly support -device=n in OpenGL mode\n");
        printf("   To use -device=n, the sample must be running w/o OpenGL\n\n");
        printf(" > %s -device=n -file=<image_name>.ppm\n", argv[0]);
        printf("exiting...\n");
        exit(EXIT_SUCCESS);
    }

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    chooseCudaDevice(argc, (const char **)argv, true); // yes to OpenGL usage

    // If the GPU does not meet SM1.1 capabilities, we quit
    if (!checkCudaCapabilities(1,1))
    {
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    // Otherwise it succeeds, we will continue to run this sample
    initData(argc, argv);

    // Initialize OpenGL context first before the CUDA context is created.  This is needed
    // to achieve optimal performance with OpenGL/CUDA interop.
    initGL(&argc, argv);
    initOpenGLBuffers(imageW, imageH);

    sdkCreateTimer(&hTimer);
    sdkStartTimer(&hTimer);

#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    setVSync(0) ;
#endif

    glutMainLoop();

    hipDeviceReset();
}
